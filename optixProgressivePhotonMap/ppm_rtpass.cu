#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "helpers.h"
#include "ppm.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(uint,     numero_hilos, , );
rtDeclareVariable(float,    scene_epsilon, , );
rtDeclareVariable(float,    ancho_intervalo, , );
rtDeclareVariable(uint,     densidad_uniforme, , );
rtDeclareVariable(uint,     umbral_bbox_hits, , );
rtDeclareVariable(float,    bbox_x, , );
rtDeclareVariable(float,    bbox_y, , );
rtDeclareVariable(float,    bbox_z, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint,     plane_number , , );
rtDeclareVariable(uint,     iteracion , , );
rtDeclareVariable(float,    seed_number , , );

rtBuffer<OriginRecord, 1>   ppass_points_buffer;
rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );

// http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
// http://richiesams.blogspot.com/2015/03/creating-randomness-and-acummulating.html

/* this GPU kernel function calculates a random number and stores it in the parameter */
static __device__ __inline__ void cuRandom(hiprandState_t& state, unsigned int& result) {

    /* hiprand works like rand - except that it takes a state as a parameter */
    result = hiprand(&state);
} // cuRandom

// regresa en &result un valor random entre [0,1]
static __device__ __inline__ void cuRandom01(hiprandState_t& state, float& result) {

  unsigned int gpu_x;
  cuRandom(state, gpu_x);
  float nrd_tmp = (float) gpu_x;
  result = fmod(nrd_tmp, 100000000.f) / 100000000.f;

} // cuRandom01

// http://mathworld.wolfram.com/SpherePointPicking.html
// http://corysimon.github.io/articles/uniformdistn-on-sphere/

// uniform sample sphere
static __device__ __inline__ void uniformSphere( const optix::float3 sample,
                                                 const optix::float3& U,
                                                 const optix::float3& V,
                                                 const optix::float3& W,
                                                 optix::float3& point )
{

    float phi   = 2.f * M_PIf * sample.x;
    float theta = acos(2.f * sample.y - 1);
    float x     = sin(theta) * cos(phi);
    float y     = sin(theta) * sin(phi);
    float z     = cos(theta);

    point = x*U + y*V + z*W;

} // uniformSphere

// https://devtalk.nvidia.com/default/topic/825579/optix/manipulating-geometry-with-keyboard-or-mouse/
// https://devtalk.nvidia.com/default/topic/853732/optix/-solved-triaid-in-optix-and-other/
/*
static __device__ __inline__ void getBboxPhoton(hiprandState_t& state, float3& o, float3& d){
  
  float sample_x, sample_y, sample_z;
  cuRandom01(state, sample_x);
  cuRandom01(state, sample_y);
  cuRandom01(state, sample_z);
  float3 sample;
  cuRandom01(state, sample.x);
  cuRandom01(state, sample.y);
  cuRandom01(state, sample.z);
  
  // genero un nuevo vector al azar en su misma recta y no mayor en modulo
  float new_x = (bbox_x*0.5f - scene_epsilon) * (2.f*sample_x - 1.f);
  float new_y = (bbox_y*0.5f - scene_epsilon) * (2.f*sample_y - 1.f);
  float new_z = (bbox_z*0.5f - scene_epsilon) * (2.f*sample_z - 1.f);

  o = make_float3(new_x, new_y, new_z);
  float3 normal = make_float3(1.f, 0.f, 0.f);

  float3 U, V, W;
  create_onb(normal, U, V, W);
  uniformSphere(sample, U, V, W, d);
  
} // getBboxPhoton
*/

static __device__ __inline__ void getPlanePhoton(hiprandState_t& state, uint intervalo, float3& o, float3& d){
  
  float sample_x, sample_y;
  cuRandom01(state, sample_x);
  cuRandom01(state, sample_y);
  float3 sample;
  cuRandom01(state, sample.x);
  cuRandom01(state, sample.y);
  cuRandom01(state, sample.z);

  // genero un nuevo vector al azar en su misma recta y no mayor en modulo
  float new_x = (bbox_x*0.5f - scene_epsilon) * (2.f*sample_x - 1.f);
  float new_y = (bbox_y*0.5f - scene_epsilon) * (2.f*sample_y - 1.f);
  float new_z = (intervalo*ancho_intervalo)-(bbox_z*0.5f - scene_epsilon)+(ancho_intervalo/2.0f);

  o = make_float3(new_x, new_y, new_z);
  float3 normal = make_float3(1.f, 0.f, 0.f);

  float3 U, V, W;
  create_onb(normal, U, V, W);
  uniformSphere(sample, U, V, W, d);
  
} // getBboxPhoton

RT_PROGRAM void rtpass_point_gen(){

  // Cada hilo se encarga de conseguir un punto dentro del sólido y colocar su coordenada en el buffer

  ppass_points_buffer[launch_index.x].origen = make_float3(0.f); // inicializo la celda que voy a calcular
  ppass_points_buffer[launch_index.x].valido = false;
  ppass_points_buffer[launch_index.x].zorder = 0u;

  float3 ray_origin, ray_direction;

  PhotonPRD prd;
  prd.temperature   = Tninguna;
  prd.trace_tmax    = RT_DEFAULT_MAX;
  prd.ray_type      = inf_ray;
  prd.pm_index      = launch_index.x;
  prd.num_deposits  = 0;
  prd.bbox_hits     = 0;
  prd.cortar_camino = 0;   // revisar mas adelante por reinicializacion
  prd.init_camino   = true;

  bool tengo_punto = false;

  hiprandState_t state;
  hiprand_init(launch_index.x+(uint)seed_number+iteracion, 0, 0, &state);

  getPlanePhoton(state, plane_number, ray_origin, ray_direction);

  prd.position      = ray_origin;
  prd.test_position = ray_origin;
  prd.direction     = ray_direction;

  uint max_intentos = 25 * umbral_bbox_hits;

  int intentos = 0;
  while (!tengo_punto && (intentos < max_intentos)){

    // dado el plano actual que vamos a procesar obtengo un punto al azar en el, y luego hago un trace para saber
    // si estoy dentro o fuera del sólido.

    ray_origin    = prd.position;
    ray_direction = prd.direction;

    optix::Ray ray = make_Ray(ray_origin, ray_direction, rtpass_ray_type, scene_epsilon, RT_DEFAULT_MAX ); 
    rtTrace( top_object, ray, prd );

    if (!prd.init_camino){ // estoy adentro
      
      tengo_punto = true;
	    ppass_points_buffer[launch_index.x].valido    = true;
	    ppass_points_buffer[launch_index.x].origen    = prd.test_position; // asigno el punto de comienzo al buffer
	    ppass_points_buffer[launch_index.x].direction = prd.direction; // asigno la dir del punto de comienzo al buffer
    
    } else if (prd.cortar_camino > 0){ // si hay error

      if (densidad_uniforme && (prd.cortar_camino == 5)){ // estoy afuera

        tengo_punto = true;
		    ppass_points_buffer[launch_index.x].valido = false;

      }else{ // si tengo que obtener un punto si o si, calculo una nueva posición
 
        getPlanePhoton(state, plane_number, ray_origin, ray_direction); 

        prd.temperature   = Tninguna;
        prd.trace_tmax    = RT_DEFAULT_MAX;
        prd.ray_type      = inf_ray;
        prd.pm_index      = launch_index.x;
        prd.num_deposits  = 0;
        prd.bbox_hits     = 0;
        prd.cortar_camino = 0;   // reinicializacion
        prd.init_camino   = true;
        prd.position      = ray_origin;
        prd.test_position = ray_origin;
        prd.direction     = ray_direction;
      }

    } // if

    intentos++;

  } // while

} // rtpass_point_gen

rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,     shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PhotonPRD,  hit_record, rtPayload, );

RT_PROGRAM void rtpass_closest_hit(){

  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float dot_product = dot(world_geometric_normal, ray.direction);

  if (dot_product > 0){ // soy un punto interno del sólido
    hit_record.init_camino = false;
  }else{ // vengo de fuera y soy un rayo test
    hit_record.cortar_camino = 5;
  } // if

} // rtpass_closest_hit
