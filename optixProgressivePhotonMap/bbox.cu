#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "ppm.h"
#include "random.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(uint,  umbral_bbox_hits, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(float, bbox_x, , );
rtDeclareVariable(float, bbox_y, , );
rtDeclareVariable(float, bbox_z, , );

//
// Closest hit material
//
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD,  hit_record,       rtPayload, );

RT_PROGRAM void bbox_closest_hit(){

  if (hit_record.bbox_hits >= umbral_bbox_hits){
    hit_record.cortar_camino = 8;
    return;
  }

  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float dot_product = dot(world_geometric_normal, ray.direction);

  float3 hit_point = ray.origin + t_hit*ray.direction;

  // si un rayo viene de afuera lo termino
  if (dot_product < 0){
    hit_record.cortar_camino = 3;
    return;
  }

  // choco con fuente
  if (abs(hit_point.z) > (bbox_z*0.5f - scene_epsilon)){ // choco con fuente, no tiro rayo

    // si es el rayo test y choco con tapa(generado dentro del solido) lo teleporto a la otra tapa para que apunte a la geometria.
    if ( hit_record.init_camino && hit_record.ray_type == inf_ray ) {
      // teleportar en z
      hit_point.z          = -hit_point.z;
    } else if (hit_point.z > (bbox_z*0.5f - scene_epsilon)){ // sino asigno las temperaturas
      hit_record.temperature = Tmax;
    } else if (hit_point.z < (-bbox_z*0.5f + scene_epsilon)){
      hit_record.temperature = Tmin;
    } // if 

  }else{

    hit_record.bbox_hits++;

    // hubo un hit con la caja, y estoy dentro, entonces discrimino el teleport del rayo
    // si la coordenada z >= emisor.z retorno temp del emisor
    // la direccion se mantiene, y solo se cambia el origen del rayo

    // cambio coordenadas del rayo
    if (abs(hit_point.y) > (bbox_y*0.5f - scene_epsilon)){

      if (hit_point.y > 0)
         hit_point.y = -hit_point.y + scene_epsilon;
      else
         hit_point.y = -hit_point.y - scene_epsilon;

    } // if y > bbox.y

    if (abs(hit_point.x) > (bbox_x*0.5f - scene_epsilon)){

      if (hit_point.x > 0)
         hit_point.x = -hit_point.x + scene_epsilon;
      else
         hit_point.x = -hit_point.x - scene_epsilon;

    } // if x > bbox.x
  } // if choco con fuente

  hit_record.position   = hit_point;
  hit_record.direction  = ray.direction;
} // bbox_closest_hit
