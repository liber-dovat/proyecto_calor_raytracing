#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "helpers.h"
#include "ppm.h"
#include "random.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(uint,     numero_hilos, , );
rtDeclareVariable(uint,     plane_number , , );
rtDeclareVariable(float,    scene_epsilon, , );
rtDeclareVariable(float,    beta_diff, , );
rtDeclareVariable(float,    delta_diff, , );
rtDeclareVariable(float,    emisividad, , );
rtDeclareVariable(float,    difusion, , );
rtDeclareVariable(uint,     densidad_uniforme, , );
rtDeclareVariable(float,    bbox_x, , );
rtDeclareVariable(float,    bbox_y, , );
rtDeclareVariable(float,    bbox_z, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(uint,     iteracion , , );
rtDeclareVariable(float,    seed_number , , );
//
// Ray generation program
//
rtBuffer<OriginRecord, 1>   ppass_points_buffer; // record :Tmin,Tmax
rtBuffer<uint, 1>           ppass_temp_buffer; // Tmin,Tmax,Tninguna
rtBuffer<uint, 1>           ppass_use_buffer; // noUsado,usado
rtBuffer<PathRecord, 1>     ppass_path_buffer; // record :Tmin,Tmax
rtDeclareVariable(uint,     num_intervalos, , );
rtDeclareVariable(float,    ancho_intervalo, , );
rtDeclareVariable(uint,     max_path_length, , );

rtDeclareVariable(uint2,    launch_index, rtLaunchIndex, );

// http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
// http://richiesams.blogspot.com/2015/03/creating-randomness-and-acummulating.html

/* this GPU kernel function calculates a random number and stores it in the parameter */
static __device__ __inline__ void cuRandom(hiprandState_t& state, unsigned int& result) {

    /* hiprand works like rand - except that it takes a state as a parameter */
    result = hiprand(&state);
} // cuRandom

// regresa en &result un valor random entre [0,1]
static __device__ __inline__ void cuRandom01(hiprandState_t& state, float& result) {

  unsigned int gpu_x;
  cuRandom(state, gpu_x);
  float nrd_tmp = (float) gpu_x;
  result = fmod(nrd_tmp, 100000000.f) / 100000000.f;

} // cuRandom01

// sample hemisphere with cosine density
static __device__ __inline__ void cosineHemisphere( const optix::float3 sample ,
                                                    const optix::float3& U,
                                                    const optix::float3& V,
                                                    const optix::float3& W,
                                                    optix::float3& point )
{

    float phi = 2.0f * M_PIf*sample.x;
    float r = (float)sqrt( sample.y );
    float x = r * (float)cos(phi);
    float y = r * (float)sin(phi);
    float z = 1.0f - sample.z;
    z = z > 0.0f ? (float)sqrt(z) : 0.0f;

    point = x*U + y*V + z*W;

} // cosineHemisphere

// uniform sample hemisphere
static __device__ __inline__ void uniformHemisphere( const optix::float3 sample,
                                                     const optix::float3& U,
                                                     const optix::float3& V,
                                                     const optix::float3& W,
                                                     optix::float3& point )
{

    float phi   = 2.f * M_PIf * sample.x;
    float theta = acos(1 - sample.y);
    float x     = sin(theta) * cos(phi);
    float y     = sin(theta) * sin(phi);
    float z     = cos(theta);

    point = x*U + y*V + z*W;
} // uniformHemisphere

// http://mathworld.wolfram.com/SpherePointPicking.html
// http://corysimon.github.io/articles/uniformdistn-on-sphere/

// uniform sample sphere
static __device__ __inline__ void uniformSphere( const optix::float3 sample,
                                                 const optix::float3& U,
                                                 const optix::float3& V,
                                                 const optix::float3& W,
                                                 optix::float3& point )
{

    float phi   = 2.f * M_PIf * sample.x;
    float theta = acos(2.f * sample.y - 1);
    float x     = sin(theta) * cos(phi);
    float y     = sin(theta) * sin(phi);
    float z     = cos(theta);

    point = x*U + y*V + z*W;

} // uniformSphere

// http://www.rorydriscoll.com/2009/01/07/better-sampling/

RT_PROGRAM void ppass_path_gen(){

  uint pm_index = launch_index.x * num_intervalos;

  float3 ray_origin, ray_direction, sample;
  float2 sample2;
  float ray_tmax;
  hiprandState_t state;
  
  if(!ppass_points_buffer[launch_index.x].valido) {
    printf("Punto %d invalido!\n", launch_index.x);  
    return;
  }
  
  hiprand_init(pm_index+(uint)seed_number+iteracion, 0, 0, &state);

  // inicializo el buffer de informacion extra de los caminos
  ppass_path_buffer[launch_index.x].caminos_generados = 0u;
  ppass_path_buffer[launch_index.x].caminos_cortados  = 0u;
  ppass_path_buffer[launch_index.x].puntos_generados  = 0u;

  // Inicializar los caminos 

  for(unsigned int i = 0; i < num_intervalos; ++i) {
    ppass_use_buffer [pm_index + i] = noUsado;
  } // for
  ppass_temp_buffer[launch_index.x] = Tninguna; 

  ppass_use_buffer[pm_index + plane_number] = usado;
  ppass_path_buffer[launch_index.x].puntos_generados++; // genero un punto    

  ray_origin = ppass_points_buffer[launch_index.x].origen; // con esto obtengo el punto de origen precalculado
  ray_direction = ppass_points_buffer[launch_index.x].direction; // con esto la direccion de origen precalculado

  PhotonPRD prd;
  prd.temperature   = Tninguna;
  prd.trace_tmax    = delta_diff;
  prd.ray_type      = delta_ray;
  prd.position      = ray_origin;
  prd.test_position = ray_origin;
  prd.direction     = ray_direction;
  prd.pm_index      = pm_index;
  prd.num_deposits  = 0;
  prd.bbox_hits     = 0;
  prd.cortar_camino = 0;   // revisar mas adelante por reinicializacion
  
  uint i_safe    = 0;
  uint num_paths = 0;
  
  bool tengo_camino = false;

  while( !tengo_camino ) {

    cuRandom01(state, sample.x);
    cuRandom01(state, sample.y);
    cuRandom01(state, sample.z);
    cuRandom01(state, sample2.x);
    cuRandom01(state, sample2.y);
    prd.sample       = sample;
    prd.rand_reflex  = sample2.x;
    prd.rand_diffuse = sample2.y;

    ray_tmax      = prd.trace_tmax;
    ray_origin    = prd.position;
    ray_direction = prd.direction;

    uint prev_num_deposits = prd.num_deposits;

    optix::Ray ray = make_Ray(ray_origin, ray_direction, ppass_and_gather_ray_type, scene_epsilon, ray_tmax ); 
    rtTrace( top_object, ray, prd );

    if (prev_num_deposits < prd.num_deposits) {
      i_safe=0;
    } // if
    i_safe++; // me aseguro de cortar el while por si num_deposits no incrementa

    if (prd.cortar_camino > 0 || i_safe > max_path_length){
      // Si cortó por return:

      tengo_camino = true;

      ppass_path_buffer[launch_index.x].caminos_cortados++;
      
    } else if (prd.temperature != Tninguna){
      // el último punto del camino llegó a una fuente

      ppass_temp_buffer[launch_index.x] = prd.temperature; 
      
      tengo_camino = true;
      num_paths++;

    } // if

  } // while

  // guardo la cantida de caminos generados
  ppass_path_buffer[launch_index.x].caminos_generados = num_paths;

  if (i_safe >= max_path_length){
    prd.cortar_camino = 7;
  }

} // ppass_path_gen

//
// Closest hit material
//
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,     shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float,      t_hit, rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD,  hit_record, rtPayload, );

static __device__ __inline__ uint getPosition(float3 point) {

  // el primer operando hace un shift de las posiciones z a valores positivos
  // el segundo es la cantida de grillas calculado como el largo dividido el ancho de los intervalos
  // luego la pos es Z>0 modulo el numero de celdas en la grilla

  float a = point.z + (bbox_z * 0.5f) - scene_epsilon;
  float b = ancho_intervalo;

  uint p = (uint)(a/b);
  
  return p;
}

static __device__ __inline__ bool cortaPlano(float3 origen, float3 destino) {
  // calcular si vector (origen,destino) corta plano de intervalo destino
  uint p_destino = getPosition(destino);
  float plano_z = (p_destino*ancho_intervalo)-(bbox_z*0.5f - scene_epsilon)+(ancho_intervalo/2.0f);
  
  return !(((origen.z - plano_z) * (destino.z - plano_z)) > 0);
}

RT_PROGRAM void ppass_closest_hit(){

  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hit_point = ray.origin + t_hit*ray.direction;
  float3 new_ray_dir;
  float  new_tmax = RT_DEFAULT_MAX;

  hit_record.bbox_hits = 0; // reinicio los golpes con la bbox

  float dot_product = dot(world_geometric_normal, ray.direction);

  // di un hit y no soy el primer rayo. Determinar si viene de dentro o de fuera
  // supongo que estoy adentro

  if (dot_product < 0 && hit_record.ray_type != inf_ray){ // si soy un rayo delta o beta externo (>0) es invalido
    hit_record.cortar_camino = 1; // corto el camino
    return;
  }

  if (dot_product >= 0 && hit_record.ray_type == inf_ray){ // si es un rayo infinito interno es invalido corto el camino
    hit_record.cortar_camino = 4;
    return;
  } // if

  bool usarSuperficie = true;
    
  if (dot_product < 0) { // vengo desde fuera

    usarSuperficie = (hit_record.rand_reflex < emisividad);

    // si no uso la superficie, reboto con coseno hacia el exterior
    if (!usarSuperficie){
      // reflejo hacia afuera
      new_tmax            = RT_DEFAULT_MAX;
      hit_record.ray_type = inf_ray;
      float3 U, V, W;
      create_onb(ffnormal, U, V, W);
      cosineHemisphere(hit_record.sample, U, V, W, new_ray_dir);
    } 
  } // if not inf_ray

  // Si uso la superficie distingo dos casos, soy delta o inf

  if (usarSuperficie) {
    bool ingreso = hit_record.rand_reflex < difusion;
    // calculo nuevo rayo segun salgo o entro

    if (ingreso){
      new_tmax            = beta_diff; // reboto usando la normal del hit_point
      hit_record.ray_type = beta_ray;

      if (dot_product < 0) { // vengo de fuera
        new_ray_dir = -ffnormal;
      }else{ // vengo de dentro
        new_ray_dir = ffnormal;
      }

    }else{
      new_tmax            = RT_DEFAULT_MAX;
      hit_record.ray_type = inf_ray;
      float3 U, V, W;

      if (dot_product < 0) { // vengo de fuera
        ffnormal = ffnormal;
      }else{ // vengo de dentro
        ffnormal = -ffnormal;
      }

      create_onb(ffnormal, U, V, W);
      cosineHemisphere(hit_record.sample, U, V, W, new_ray_dir);
    } // if ingreso

  } //if usarSuperficie

  // guardo info del choque
  if(cortaPlano(ray.origin, hit_point) && (dot_product >= 0)) {
    ppass_use_buffer[hit_record.pm_index + getPosition(hit_point)] = usado;
    ppass_path_buffer[launch_index.x].puntos_generados++; // genero un punto
  }

  hit_record.num_deposits++;

  // si me paso del numero de fotones del camino reinicio camino
  if ( hit_record.num_deposits >= max_path_length ){
    hit_record.cortar_camino = 6;
    return;
  }

  hit_record.position   = hit_point;
  hit_record.direction  = new_ray_dir;
  hit_record.trace_tmax = new_tmax;

} // ppass_closest_hit2

// miss
// si ray.tmax == INIFNITY retorno
// sino 
// estoy adentro, guardo punto, calculo otra direccion y tiro otro rayo con tmax=delta_diff.

RT_PROGRAM void ppass_miss(){

  // si el rayo se iba a infinito, es decir, salía del objeto y dio miss con la escena, termino y retorno
  if (hit_record.ray_type == inf_ray){
    hit_record.cortar_camino = 2;
    return;
  } else if (hit_record.ray_type == delta_ray || hit_record.ray_type == beta_ray){ // si venía con vector delta_diff y dio miss estoy adentro

    // calculo el hit point como el origen mas delta/beta segun energy.z
    float tmaxTemp = inf_ray;
    if (hit_record.ray_type == delta_ray ){
      tmaxTemp = delta_diff;
    }else if (hit_record.ray_type == beta_ray ){
      tmaxTemp = beta_diff;
    } // if
        
    float3 new_coord = ray.origin + (tmaxTemp - scene_epsilon) * ray.direction;
    float3 new_ray_dir;

    float3 U, V, W;
    create_onb(ray.direction, U, V, W);
    uniformSphere(hit_record.sample, U, V, W, new_ray_dir);

    // guardo punto
    if(cortaPlano(ray.origin, new_coord)) {
      ppass_use_buffer[hit_record.pm_index + getPosition(new_coord)] = usado;
      ppass_path_buffer[launch_index.x].puntos_generados++; // genero un punto      
    }

    hit_record.ray_type = delta_ray;
    hit_record.num_deposits++;

    // si me paso del numero de fotones del camino reinicio camino
    if ( hit_record.num_deposits >= max_path_length){
      hit_record.cortar_camino = 6;
      return;
    }

    hit_record.position   = new_coord;
    hit_record.direction  = new_ray_dir;
    hit_record.trace_tmax = delta_diff;

  } // if ray.tmax

} // rtpass_miss2
